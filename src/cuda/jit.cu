#include <hip/hip_runtime.h>

#include <misaki/utils/cuda/common.cuh>
#include <sstream>

namespace misaki {
namespace cuda {

char* msk_cuda_whos() {
  std::ostringstream oss;
  oss << std::endl;
  oss << "=========================================" << std::endl;
  return strdup(oss.str().c_str());
}

}  // namespace cuda
}  // namespace misaki
